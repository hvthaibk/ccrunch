/***********************************************************************
 *  File:       cufft_wrapper.cu
 *
 *  Purpose:    Implementation of cufft-related functions
 *
 *  Author:     Thai V. Hoang
 *
 *  Contact:    hvthaibk@gmail.com
 *
 *  Copyright (C) 2012 Thai V. Hoang, INRIA
 **********************************************************************/

#include "cufft_wrapper.cuh"

namespace gem {

/*****************************************
 * Forward and inverse FFTs (slow)
 ****************************************/

// 2D float: forward
void cufft_dft_r2c(float          *dataReal, size_t nRowReal, size_t nColReal,
                   hipFloatComplex *dataCplx)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan2d(&fftPlan, (int) nRowReal, (int) nColReal, HIPFFT_R2C) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecR2C(fftPlan, (hipfftReal *)    dataReal,
                                           (hipfftComplex *) dataCplx) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 2D float: inverse
void cufft_dft_c2r(hipFloatComplex *dataCplx,
                   float          *dataReal, size_t nRowReal, size_t nColReal)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan2d(&fftPlan, (int) nRowReal, (int) nColReal, HIPFFT_C2R) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecC2R(fftPlan, (hipfftComplex *) dataCplx,
                                           (hipfftReal *)    dataReal) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 2D double: forward
void cufft_dft_r2c(double          *dataReal, size_t nRowReal, size_t nColReal,
                   hipDoubleComplex *dataCplx)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan2d(&fftPlan, (int) nRowReal, (int) nColReal, HIPFFT_D2Z) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecD2Z(fftPlan, (hipfftDoubleReal *)    dataReal,
                                           (hipfftDoubleComplex *) dataCplx) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 2D double: inverse
void cufft_dft_c2r(hipDoubleComplex *dataCplx,
                   double          *dataReal, size_t nRowReal, size_t nColReal)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan2d(&fftPlan, (int) nRowReal, (int) nColReal, HIPFFT_D2Z) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecZ2D(fftPlan, (hipfftDoubleComplex *) dataCplx,
                                           (hipfftDoubleReal *)    dataReal) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 3D float: forward
void cufft_dft_r2c(float          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal,
                   hipFloatComplex *dataCplx)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0 && nSecReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan3d(&fftPlan, (int) nRowReal, (int) nColReal, (int) nSecReal, HIPFFT_R2C) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecR2C(fftPlan, (hipfftReal *)    dataReal,
                                           (hipfftComplex *) dataCplx) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 3D float: inverse
void cufft_dft_c2r(hipFloatComplex *dataCplx,
                   float          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0 && nSecReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan3d(&fftPlan, (int) nRowReal, (int) nColReal, (int) nSecReal, HIPFFT_C2R) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecC2R(fftPlan, (hipfftComplex *) dataCplx,
                                           (hipfftReal *)    dataReal) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 3D double: forward
void cufft_dft_r2c(double          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal,
                   hipDoubleComplex *dataCplx)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0 && nSecReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan3d(&fftPlan, (int) nRowReal, (int) nColReal, (int) nSecReal, HIPFFT_D2Z) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecD2Z(fftPlan, (hipfftDoubleReal *)    dataReal,
                                           (hipfftDoubleComplex *) dataCplx) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

// 3D double: inverse
void cufft_dft_c2r(hipDoubleComplex *dataCplx,
                   double          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0 && nSecReal > 0);

    // setup CUFFT plan
    hipfftHandle     fftPlan;
    CUFFT_SAFE_CALL( hipfftPlan3d(&fftPlan, (int) nRowReal, (int) nColReal, (int) nSecReal, HIPFFT_D2Z) );

    // obtain the FFT
    CUFFT_SAFE_CALL( hipfftExecZ2D(fftPlan, (hipfftDoubleComplex *) dataCplx,
                                           (hipfftDoubleReal *)    dataReal) );

    // deallocate CUFFT plan
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan) );
}

/*****************************************
 * Forward and inverse FFTs (1d1d1d)
 ****************************************/

// 2D float: forward
void cufft_dft_r2c_fast(float          *dataReal, size_t nRowReal, size_t nColReal,
                        hipFloatComplex *dataCplx)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0);

    size_t    nColCplx = nColReal / 2 + 1;
    size_t    fftSize  = nRowReal * nColCplx;

    // allocate CUFFT input and output arrays
    hipFloatComplex    *dataTmp1 = NULL;
    cuda_arrayDev_new(dataTmp1, fftSize);

    // setup CUFFT plans
    hipfftHandle fftPlan1, fftPlan2;
    CUFFT_SAFE_CALL( hipfftPlan1d(&fftPlan1, (int) nColReal, HIPFFT_R2C, (int) nRowReal) );
    CUFFT_SAFE_CALL( hipfftPlan1d(&fftPlan2, (int) nRowReal, HIPFFT_C2C, (int) nColCplx) );

    // batch FFT 1
    CUFFT_SAFE_CALL( hipfftExecR2C(fftPlan1, (hipfftReal *)    dataReal,
                                            (hipfftComplex *) dataTmp1) );

    // permute 1
    cuda_array_permute(reinterpret_cast<hipFloatComplex*>(dataTmp1),
                       reinterpret_cast<hipFloatComplex*>(dataCplx),
                       nRowReal, nColCplx);

    // batch FFT 2
    CUFFT_SAFE_CALL( hipfftExecC2C(fftPlan2, (hipfftComplex *) dataCplx,
                                            (hipfftComplex *) dataTmp1,
                                            HIPFFT_FORWARD) );

    // permute 2
    cuda_array_permute(reinterpret_cast<hipFloatComplex*>(dataTmp1),
                       reinterpret_cast<hipFloatComplex*>(dataCplx),
                       nColCplx, nRowReal);

    // deallocate CUFFT arrays and plans
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan1) );
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan2) );
    cuda_arrayDev_delete(dataTmp1);
}

// 2D float: forward
/*void cufft_dft_r2c_fast(float          *dataReal, size_t nRowReal, size_t nColReal,
                        hipFloatComplex *dataCplx)
{
    assert(dataReal != NULL);
    assert(dataCplx != NULL);
    assert(nRowReal > 0 && nColReal > 0);

    size_t    nColCplx = nColReal / 2 + 1;
    size_t    fftSize  = nRowReal * nColCplx;

    // allocate CUFFT input and output arrays
    hipFloatComplex    *dataTmp1 = NULL;
    cuda_arrayDev_new(dataTmp1, fftSize);

    // setup CUFFT plans
    int         n1[] = {(int) nColReal};
    int         n2[] = {(int) nRowReal};
    hipfftHandle fftPlan1, fftPlan2;
    CUFFT_SAFE_CALL( hipfftPlanMany(&fftPlan1, (int) 1,  n1,
                                   n1, (int) 1, (int) nColReal,
                                   n1, (int) 1, (int) nColCplx,
                                   HIPFFT_R2C, (int) nRowReal) );
    CUFFT_SAFE_CALL( hipfftPlanMany(&fftPlan2, (int) 1,  n2,
                                   n2, (int) nColCplx, (int) 1,
                                   n2, (int) nColCplx, (int) 1,
                                   HIPFFT_C2C, (int) nColCplx) );
    //CUFFT_SAFE_CALL( hipfftPlanMany(&fftPlan2, (int) 1,  n2,
    //                               n2, (int) 1, (int) nRowReal,
    //                               n2, (int) 1, (int) nRowReal,
    //                               HIPFFT_C2C, (int) nColCplx) );

    // batch FFT 1
    CUFFT_SAFE_CALL( hipfftExecR2C(fftPlan1, (hipfftReal *)    dataReal,
                                            (hipfftComplex *) dataTmp1) );

    // batch FFT 2
    CUFFT_SAFE_CALL( hipfftExecC2C(fftPlan2, (hipfftComplex *) dataCplx,
                                            (hipfftComplex *) dataTmp1,
                                            HIPFFT_FORWARD) );

    // deallocate CUFFT arrays and plans
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan1) );
    CUFFT_SAFE_CALL( hipfftDestroy(fftPlan2) );
    cuda_arrayDev_delete(dataTmp1);
}*/

// 2D float: inverse
void cufft_dft_c2r_fast(hipFloatComplex *dataCplx,
                        float          *dataReal, size_t nRowReal, size_t nColReal);

// 2D double: forward
void cufft_dft_r2c_fast(double          *dataReal, size_t nRowReal, size_t nColReal,
                        hipDoubleComplex *dataCplx);

// 2D double: inverse
void cufft_dft_c2r_fast(hipDoubleComplex *dataCplx,
                        double          *dataReal, size_t nRowReal, size_t nColReal);

// 3D float: forward
void cufft_dft_r2c_fast(float          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal,
                        hipFloatComplex *dataCplx);

// 3D float: inverse
void cufft_dft_c2r_fast(hipFloatComplex *dataCplx,
                        float          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal);

// 3D double: forward
void cufft_dft_r2c_fast(double          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal,
                        hipDoubleComplex *dataCplx);

// 3D double: inverse
void cufft_dft_c2r_fast(hipDoubleComplex *dataCplx,
                        double          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal);

/*****************************************
 * Forward and inverse FFTs (2d1d)
 ****************************************/

// 3D float
void cufft_dft_r2c_2d1d(float          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal,
                        hipFloatComplex *dataCplx);

void cufft_dft_c2r_2d1d(hipFloatComplex *dataCplx,
                        float          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal);

// 3D double
void cufft_dft_r2c_2d1d(double          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal,
                        hipDoubleComplex *dataCplx);

void cufft_dft_c2r_2d1d(hipDoubleComplex *dataCplx,
                        double          *dataReal, size_t nRowReal, size_t nColReal, size_t nSecReal);

} // namespace gem
