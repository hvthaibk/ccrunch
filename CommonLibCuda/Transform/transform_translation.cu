#include "hip/hip_runtime.h"
/***********************************************************************
 *  File:       transform_translation.cu
 *
 *  Purpose:    Implementation of transformation functions
 *
 *  Author:     Thai V. Hoang
 *
 *  Contact:    hvthaibk@gmail.com
 *
 *  Copyright (C) 2012 Thai V. Hoang, INRIA
 **********************************************************************/

#include "transform.cuh"

namespace gem {

/**************************
 * Translation
 *************************/

// 1D
template <typename T> __global__
void dev_transform_translate(const T* const arraySrc, T* const arrayDst,
                             size_t nRow,
                             T      nRowOff)
{
    size_t    iRow = blockDim.x * blockIdx.x + threadIdx.x;

    ptrdiff_t    intRow, intRow1;
    bool         bRow, bRow1;
    T            iRowOri;
    T            v0, v1;
    T            fracRow;

    if (iRow < nRow) {
        iRowOri = (T) iRow - nRowOff;

        intRow = (ptrdiff_t) std::floor(iRowOri);

        intRow1 = intRow + 1;

        bRow  = (intRow  >= 0) && (intRow  <= (ptrdiff_t) nRow-1);
        bRow1 = (intRow1 >= 0) && (intRow1 <= (ptrdiff_t) nRow-1);

        v0 = 0;     v1 = 0;

        if (bRow )  v0 = arraySrc[intRow ];
        if (bRow1)  v1 = arraySrc[intRow1];

        fracRow = iRowOri - (T) intRow;

        arrayDst[iRow] = (1-fracRow) * v0 + fracRow * v1;
    }
}

template <typename T>
void cuda_transform_translate(const T* const arraySrc, T* const arrayDst,
                              size_t nRow,
                              T      nRowOff,
                              eInter inter)
{
    assert(arraySrc != NULL && arrayDst != NULL);
    assert(nRow > 0);

    switch (inter) {
        case INTER_NEAREST:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
            break;
        case INTER_LINEAR:
            break;
        case INTER_CUBIC:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
            break;
        default:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
    }

    dev_transform_translate<<<iDivUp(nRow, BLOCK_1D_NROW), BLOCK_1D_NROW>>>
        (arraySrc, arrayDst,
         nRow,
         nRowOff);
}

// instantiation
template
void cuda_transform_translate<float >(const float*  const arraySrc, float * const arrayDst,
                                      size_t nRow,
                                      float  nRowOff,
                                      eInter inter);
template
void cuda_transform_translate<double>(const double* const arraySrc, double* const arrayDst,
                                      size_t nRow,
                                      double nRowOff,
                                      eInter inter);

// 2D
template <typename T> __global__
void dev_transform_translate(const T* const arraySrc, T* const arrayDst,
                             size_t nRow,    size_t nCol,
                             T      nRowOff, T      nColOff)
{
    size_t    iRow = blockDim.y * blockIdx.y + threadIdx.y;
    size_t    iCol = blockDim.x * blockIdx.x + threadIdx.x;

    ptrdiff_t    intRow, intCol, intRow1, intCol1;
    bool         bRow, bCol, bRow1, bCol1;
    T            iRowOri, iColOri;
    T            v00, v01, v10, v11;
    T            fracRow, fracCol;

    if (iRow < nRow && iCol < nCol) {
        iRowOri = (T) iRow - nRowOff;
        iColOri = (T) iCol - nColOff;

        intRow = (ptrdiff_t) std::floor(iRowOri);
        intCol = (ptrdiff_t) std::floor(iColOri);

        intRow1 = intRow + 1;
        intCol1 = intCol + 1;

        bRow  = (intRow  >= 0) && (intRow  <= (ptrdiff_t) nRow-1);
        bCol  = (intCol  >= 0) && (intCol  <= (ptrdiff_t) nCol-1);
        bRow1 = (intRow1 >= 0) && (intRow1 <= (ptrdiff_t) nRow-1);
        bCol1 = (intCol1 >= 0) && (intCol1 <= (ptrdiff_t) nCol-1);

        v00 = 0;    v01 = 0;    v10 = 0;    v11 = 0;

        if (bRow  && bCol ) v00 = arraySrc[intRow *nCol+intCol ];
        if (bRow  && bCol1) v01 = arraySrc[intRow *nCol+intCol1];
        if (bRow1 && bCol ) v10 = arraySrc[intRow1*nCol+intCol ];
        if (bRow1 && bCol1) v11 = arraySrc[intRow1*nCol+intCol1];

        fracRow = iRowOri - (T) intRow;
        fracCol = iColOri - (T) intCol;

        arrayDst[iRow*nCol+iCol] = (1-fracRow) * (1-fracCol) * v00 +
                                   (1-fracRow) * fracCol     * v01 +
                                   fracRow     * (1-fracCol) * v10 +
                                   fracRow     * fracCol     * v11;
    }
}

template <typename T>
void cuda_transform_translate(const T* const arraySrc, T* const arrayDst,
                              size_t nRow,    size_t nCol,
                              T      nRowOff, T      nColOff,
                              eInter inter)
{
    assert(arraySrc != NULL && arrayDst != NULL);
    assert(nRow > 0 && nCol > 0);

    switch (inter) {
        case INTER_NEAREST:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
            break;
        case INTER_LINEAR:
            break;
        case INTER_CUBIC:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
            break;
        default:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
    }

    dim3    dimBlock(BLOCK_2D_NCOL, BLOCK_2D_NROW);
    dim3    dimGrid (iDivUp(nCol, dimBlock.x),
                     iDivUp(nRow, dimBlock.y));

    dev_transform_translate<<<dimGrid, dimBlock>>>
        (arraySrc, arrayDst,
         nRow,    nCol,
         nRowOff, nColOff);
}

// instantiation
template
void cuda_transform_translate<float >(const float*  const arraySrc, float * const arrayDst,
                                      size_t nRow,    size_t nCol,
                                      float  nRowOff, float  nColOff,
                                      eInter inter);
template
void cuda_transform_translate<double>(const double* const arraySrc, double* const arrayDst,
                                      size_t nRow,    size_t nCol,
                                      double nRowOff, double nColOff,
                                      eInter inter);

// 3D
template <typename T> __global__
void dev_transform_translate(const T* const arraySrc, T* const arrayDst,
                             size_t nRow,    size_t nCol,    size_t nSec,
                             T      nRowOff, T      nColOff, T      nSecOff)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#else
    size_t    iRow = blockDim.z * blockIdx.z + threadIdx.z;
    size_t    iCol = blockDim.y * blockIdx.y + threadIdx.y;
    size_t    iSec = blockDim.x * blockIdx.x + threadIdx.x;

    ptrdiff_t    intRow, intCol, intSec, intRow1, intCol1, intSec1;
    bool         bRow, bCol, bSec, bRow1, bCol1, bSec1;
    T            iRowOri, iColOri, iSecOri;
    T            v000, v001, v010, v011, v100, v101, v110, v111;
    T            fracRow, fracCol, fracSec;

    if (iRow < nRow && iCol < nCol && iSec <nSec) {
        iRowOri = (T) iRow - nRowOff;
        iColOri = (T) iCol - nColOff;
        iSecOri = (T) iSec - nSecOff;

        intRow = (ptrdiff_t) std::floor(iRowOri);
        intCol = (ptrdiff_t) std::floor(iColOri);
        intSec = (ptrdiff_t) std::floor(iSecOri);

        intRow1 = intRow + 1;
        intCol1 = intCol + 1;
        intSec1 = intSec + 1;

        bRow  = (intRow  >= 0) && (intRow  <= (ptrdiff_t) nRow-1);
        bCol  = (intCol  >= 0) && (intCol  <= (ptrdiff_t) nCol-1);
        bSec  = (intSec  >= 0) && (intSec  <= (ptrdiff_t) nSec-1);
        bRow1 = (intRow1 >= 0) && (intRow1 <= (ptrdiff_t) nRow-1);
        bCol1 = (intCol1 >= 0) && (intCol1 <= (ptrdiff_t) nCol-1);
        bSec1 = (intSec1 >= 0) && (intSec1 <= (ptrdiff_t) nSec-1);

        v000 = 0;    v001 = 0;    v010 = 0;    v011 = 0;
        v100 = 0;    v101 = 0;    v110 = 0;    v111 = 0;

        if (bRow  && bCol  && bSec ) v000 = arraySrc[intRow *nCol*nSec+intCol *nSec+intSec ];
        if (bRow  && bCol  && bSec1) v001 = arraySrc[intRow *nCol*nSec+intCol *nSec+intSec1];
        if (bRow  && bCol1 && bSec ) v010 = arraySrc[intRow *nCol*nSec+intCol1*nSec+intSec ];
        if (bRow  && bCol1 && bSec1) v011 = arraySrc[intRow *nCol*nSec+intCol1*nSec+intSec1];
        if (bRow1 && bCol  && bSec ) v100 = arraySrc[intRow1*nCol*nSec+intCol *nSec+intSec ];
        if (bRow1 && bCol  && bSec1) v101 = arraySrc[intRow1*nCol*nSec+intCol *nSec+intSec1];
        if (bRow1 && bCol1 && bSec ) v110 = arraySrc[intRow1*nCol*nSec+intCol1*nSec+intSec ];
        if (bRow1 && bCol1 && bSec1) v111 = arraySrc[intRow1*nCol*nSec+intCol1*nSec+intSec1];

        fracRow = iRowOri - (T) intRow;
        fracCol = iColOri - (T) intCol;
        fracSec = iSecOri - (T) intSec;

        arrayDst[(iRow*nCol+iCol)*nSec+iSec]
                    = (1-fracRow) * (1-fracCol) * (1-fracSec) * v000
                    + (1-fracRow) * (1-fracCol) * fracSec     * v001
                    + (1-fracRow) * fracCol     * (1-fracSec) * v010
                    + (1-fracRow) * fracCol     * fracSec     * v011
                    + fracRow     * (1-fracCol) * (1-fracSec) * v100
                    + fracRow     * (1-fracCol) * fracSec     * v101
                    + fracRow     * fracCol     * (1-fracSec) * v110
                    + fracRow     * fracCol     * fracSec     * v111;
    }
#endif
}

template <typename T>
void cuda_transform_translate(const T* const arraySrc, T* const arrayDst,
                              size_t nRow,    size_t nCol,    size_t nSec,
                              T      nRowOff, T      nColOff, T      nSecOff,
                              eInter inter)
{
    assert(arraySrc != NULL && arrayDst != NULL);
    assert(nRow > 0 && nCol > 0 && nSec > 0);

    switch (inter) {
        case INTER_NEAREST:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
            break;
        case INTER_LINEAR:
            break;
        case INTER_CUBIC:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
            break;
        default:
            ERROR("cuda_transform_translate", "unsupported interpolation mode");
    }

#ifdef __GEM_CUDA_ARCH_HOST_130__
    ERROR("cuda_transform_translate", "unsupported hardware");
    /*dim3    dimBlock(BLOCK_2D_NCOL, BLOCK_2D_NROW);
    dim3    dimGrid (iDivUp(nSec, dimBlock.x),
                     iDivUp(nCol, dimBlock.y));*/
#else
    dim3    dimBlock(BLOCK_3D_NSEC, BLOCK_3D_NCOL, BLOCK_3D_NROW);
    dim3    dimGrid (iDivUp(nSec, dimBlock.x),
                     iDivUp(nCol, dimBlock.y),
                     iDivUp(nRow, dimBlock.z));
#endif

    dev_transform_translate<<<dimGrid, dimBlock>>>
        (arraySrc, arrayDst,
         nRow,    nCol,    nSec,
         nRowOff, nColOff, nSecOff);
}

// instantiation
template
void cuda_transform_translate<float >(const float*  const arraySrc, float * const arrayDst,
                                      size_t nRow,    size_t nCol,    size_t nSec,
                                      float  nRowOff, float  nColOff, float  nSecOff,
                                      eInter inter);
template
void cuda_transform_translate<double>(const double* const arraySrc, double* const arrayDst,
                                      size_t nRow,    size_t nCol,    size_t nSec,
                                      double nRowOff, double nColOff, double nSecOff,
                                      eInter inter);

} // namespace gem
